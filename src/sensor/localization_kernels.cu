#include "hip/hip_runtime.h"

// CUDA Dependencies
#include <hip/hip_runtime.h>

// Thrust Dependencies
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/reduce.h>

// Octree-SLAM Dependencies
#include <octree_slam/sensor/localization_kernels.h>

namespace octree_slam {

namespace sensor {

__device__ const float DIST_THRESH = 0.1f; //Use 10 cm distance threshold for correspondences
__device__ const float NORM_THRESH = 0.87f; //Use 30 degree orientation threshold

//Define structure to be used for combined Mat6x6 and Vec6 in thrust summation
struct Mat6x7 {
  float values[42];
  __host__ __device__ Mat6x7() {};
  __host__ __device__ Mat6x7(const int val) {
    for (int i = 0; i < 42; i++) {
      values[i] = val;
    }
  };
};

__host__ __device__ inline Mat6x7 operator+(const Mat6x7& lhs, const Mat6x7& rhs) {
  Mat6x7 result;
  for (int i = 0; i < 42; i++) {
    result.values[i] = lhs.values[i] + rhs.values[i];
  }
  return result;
}

ICPFrame::ICPFrame(const int w, const int h) : width(w), height(h) {
  hipMalloc((void**)&vertex, width*height*sizeof(glm::vec3));
  hipMalloc((void**)&normal, width*height*sizeof(glm::vec3));
};

ICPFrame::~ICPFrame() {
  hipFree(vertex);
  hipFree(normal);
}

RGBDFrame::RGBDFrame(const int w, const int h) : width(w), height(h) {
  hipMalloc((void**)&intensity, width*height*sizeof(float));
  hipMalloc((void**)&vertex, width*height*sizeof(glm::vec3));
}

RGBDFrame::~RGBDFrame() {
  hipFree(intensity);
  hipFree(vertex);
}

__global__ void computeICPCorrespondences(const glm::vec3* last_frame_vertex, const glm::vec3* last_frame_normal, const glm::vec3* this_frame_vertex, const glm::vec3* this_frame_normal, 
    const int num_points, bool* stencil, int* num_corr) {

  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  //Don't do anything if the index is out of bounds
  if (idx >= num_points) {
    return;
  }

  bool is_match = true;

  //Check whether points are any good
  if (!isfinite(this_frame_vertex[idx].x) || !isfinite(this_frame_vertex[idx].y) || !isfinite(this_frame_vertex[idx].z)
    || !isfinite(last_frame_vertex[idx].x) || !isfinite(last_frame_vertex[idx].y) || !isfinite(last_frame_vertex[idx].z)) {
    is_match = false;
  }
  if (!is_match || !isfinite(this_frame_normal[idx].x) || !isfinite(this_frame_normal[idx].y) || !isfinite(this_frame_normal[idx].z)
    || !isfinite(last_frame_normal[idx].x) || !isfinite(last_frame_normal[idx].y) || !isfinite(last_frame_normal[idx].z)) {
    is_match = false;
  }

  //Check position difference
  if (!is_match || glm::length(this_frame_vertex[idx] - last_frame_vertex[idx]) > DIST_THRESH) {
    is_match = false;
  }

  //Check normal difference
  if (!is_match || glm::dot(this_frame_normal[idx], last_frame_normal[idx]) < NORM_THRESH) {
    is_match = false;
  }

  //Update result
  stencil[idx] = is_match;

  //Subtract from global counter if its not a match
  if (!is_match) {
    atomicAdd(num_corr, -1);
  }

}

__global__ void computeICPCostsKernel(const glm::vec3* last_frame_normal, const glm::vec3* last_frame_vertex, const glm::vec3* this_frame_vertex, const int num_points, const int load_size, Mat6x7* As) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  //Don't do anything if the index is out of bounds
  if (idx*load_size >= num_points) {
    return;
  }

  //Determine whether the full load is in the bounds
  int bound = load_size;
  if ((idx + 1)*load_size - num_points > 0) {
    bound -= (idx + 1)*load_size - num_points;
  }

  //Init outputs
  for (int i = 0; i < 7; i++) {
    for (int j = 0; j < 6; j++) {
      As[idx].values[6 * i + j] = 0.0f;
    }
  }

  //Loop through the load
  for (int k = 0; k < bound; k++) {

    //Get the vertex and normal values
    glm::vec3 v2 = this_frame_vertex[load_size*idx+k];
    glm::vec3 v1 = last_frame_vertex[load_size*idx+k];
    glm::vec3 n = last_frame_normal[load_size*idx+k];

    //Construct A_T
    float G_T[18] = { 0.0f, -v2.x, -v2.y, -v2.z, 0.0f, v2.x, v2.y, v2.z, 0.0f,
      1.0f, 0.0f, 0.0f, 0.0f, 1.0f, 0.0f, 0.0f, 0.0f, 1.0f };
    float A_T[6];
    for (int i = 0; i < 6; i++) {
      A_T[i] = G_T[3 * i] * n.x + G_T[3*i + 1] * n.y + G_T[3*i + 2] * n.z;
    }

    //Construct b
    float b = glm::dot(n, v1 - v2);

    //Compute outputs
    for (int i = 0; i < 6; i++) {
      for (int j = 0; j < 6; j++) {
        As[idx].values[6*i + j] += A_T[i] * A_T[j];
      }
    }
    for (int i = 0; i < 6; i++) {
      As[idx].values[36 + i] += b*A_T[i];
    }

  }
}

__global__ void computeICPCostsUncorrespondedKernel(const glm::vec3* last_frame_normal, const glm::vec3* last_frame_vertex, const glm::vec3* this_frame_normal, 
  const glm::vec3* this_frame_vertex, const int num_points, const int load_size, Mat6x7* As) {

  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  //Don't do anything if the index is out of bounds
  if (idx*load_size >= num_points) {
    return;
  }

  //Determine whether the full load is in the bounds
  int bound = load_size;
  if ((idx + 1)*load_size - num_points > 0) {
    bound -= (idx + 1)*load_size - num_points;
  }

  //Init outputs
  for (int i = 0; i < 7; i++) {
    for (int j = 0; j < 6; j++) {
      As[idx].values[6 * i + j] = 0.0f;
    }
  }

  //Loop through the load
  for (int k = 0; k < bound; k++) {

    //Get the vertex and normal values
    glm::vec3 v2 = this_frame_vertex[load_size*idx+k];
    glm::vec3 n2 = this_frame_normal[load_size*idx+k];
    glm::vec3 v1 = last_frame_vertex[load_size*idx+k];
    glm::vec3 n1 = last_frame_normal[load_size*idx+k];

    //Check whether points are any good
    if (!isfinite(v2.x) || !isfinite(v2.y) || !isfinite(v2.z)
      || !isfinite(v1.x) || !isfinite(v1.y) || !isfinite(v1.z)
      || (v1.z < 0.1f) || (v2.z < 0.1f) || (v1.z > 10.0f) || (v2.z > 10.0f)) {
      continue;
    }
    if (!isfinite(n2.x) || !isfinite(n2.y) || !isfinite(n2.z)
      || !isfinite(n1.x) || !isfinite(n1.y) || !isfinite(n1.z)) {
      continue;
    }

    //Check position difference
    if (glm::length(v2 - v1) > DIST_THRESH) {
      continue;
    }

    //Check normal difference
    if (glm::dot(n2, n1) < NORM_THRESH) {
      continue;
    }

    //Construct A_T
    float G_T[18] = { 0.0f, -v2.x, -v2.y, -v2.z, 0.0f, v2.x, v2.y, v2.z, 0.0f,
      1.0f, 0.0f, 0.0f, 0.0f, 1.0f, 0.0f, 0.0f, 0.0f, 1.0f };
    float A_T[6];
    for (int i = 0; i < 6; i++) {
      A_T[i] = G_T[3 * i] * n1.x + G_T[3 * i + 1] * n1.y + G_T[3 * i + 2] * n1.z;
    }

    //Construct b
    float b = glm::dot(n1, v1 - v2);

    //Compute outputs
    for (int i = 0; i < 6; i++) {
      for (int j = 0; j < 6; j++) {
        As[idx].values[6 * i + j] += A_T[i] * A_T[j];
      }
    }
    for (int i = 0; i < 6; i++) {
      As[idx].values[36+i] += b*A_T[i];
    }

  }
}

extern "C" void computeICPCost(const ICPFrame* last_frame, const ICPFrame &this_frame, float* A, float* b) {
  //TODO: Verify that the two frames are the same size

  //Compute correspondences 
  int num_correspondences = this_frame.width * this_frame.height;
  bool* d_stencil;
  int* d_num_corr;
  hipMalloc((void**)&d_stencil, num_correspondences * sizeof(bool));
  hipMalloc((void**)&d_num_corr, sizeof(int));
  hipMemcpy(d_num_corr, &num_correspondences, sizeof(int), hipMemcpyHostToDevice); //Initialize to the total points. Assume that most points will be valid
  computeICPCorrespondences<<<ceil((float)num_correspondences /256.0f), 256>>>(last_frame->vertex, last_frame->normal, this_frame.vertex, this_frame.normal, 
    num_correspondences, d_stencil, d_num_corr);
  hipDeviceSynchronize();

  //Copy number of correspondences back from the device
  hipMemcpy(&num_correspondences, d_num_corr, sizeof(int), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  hipFree(d_num_corr);

  //Don't continue without any correspondences
  if (num_correspondences <= 0) {
    return;
  }

  //Allocate memory for reduced copies
  glm::vec3* last_frame_reduced_vertex;
  hipMalloc((void**)&last_frame_reduced_vertex, num_correspondences * sizeof(glm::vec3));
  glm::vec3* last_frame_reduced_normal;
  hipMalloc((void**)&last_frame_reduced_normal, num_correspondences * sizeof(glm::vec3));
  glm::vec3* this_frame_reduced_vertex;
  hipMalloc((void**)&this_frame_reduced_vertex, num_correspondences * sizeof(glm::vec3));

  //Reduce inputs with thrust compaction
  thrust::device_ptr<glm::vec3> in, out;
  thrust::device_ptr<bool> sten = thrust::device_pointer_cast<bool>(d_stencil);
  in = thrust::device_pointer_cast<glm::vec3>(last_frame->vertex);
  out = thrust::device_pointer_cast<glm::vec3>(last_frame_reduced_vertex);
  thrust::copy_if(in, in + last_frame->width*last_frame->height, sten, out, thrust::identity<bool>());
  in = thrust::device_pointer_cast<glm::vec3>(last_frame->normal);
  out = thrust::device_pointer_cast<glm::vec3>(last_frame_reduced_normal);
  thrust::copy_if(in, in + last_frame->width*last_frame->height, sten, out, thrust::identity<bool>());
  in = thrust::device_pointer_cast<glm::vec3>(this_frame.vertex);
  out = thrust::device_pointer_cast<glm::vec3>(this_frame_reduced_vertex);
  thrust::copy_if(in, in + last_frame->width*last_frame->height, sten, out, thrust::identity<bool>());
  
  //Free device memory from data in the compaction stages
  hipFree(d_stencil);

  //Compute cost terms
  int load_size = 10;
  Mat6x7* d_A; //Note, the 6x6 A and 6x1 b are combined into a single array so they can be reduced together with thrust later
  hipMalloc((void**) &d_A, (num_correspondences/load_size) * sizeof(Mat6x7));
  computeICPCostsKernel<<<ceil((float)num_correspondences / 16.0f / (float)load_size), 16>>>(last_frame_reduced_normal, last_frame_reduced_vertex, this_frame_reduced_vertex, num_correspondences, load_size, d_A);
  hipDeviceSynchronize();

  //Free up device memory
  hipFree(last_frame_reduced_vertex);
  hipFree(last_frame_reduced_normal);
  hipFree(this_frame_reduced_vertex);

  //Sum terms (reduce) with thrust
  thrust::device_ptr<Mat6x7> thrust_A = thrust::device_pointer_cast<Mat6x7>(d_A);
  Mat6x7 matA = thrust::reduce(thrust_A, thrust_A + (num_correspondences/load_size));

  //Free up device memory
  hipFree(d_A);

  //Copy result to output
  memcpy(A, matA.values, 36 * sizeof(float));
  memcpy(b, matA.values + 36, 6 * sizeof(float));
}

extern "C" void computeICPCost2(const ICPFrame* last_frame, const ICPFrame &this_frame, float* A, float* b) {
  //TODO: Verify that the two frames are the same size

  //Assume all are correspondences 
  int num_correspondences = this_frame.width * this_frame.height;

  //Compute cost terms
  int load_size = 20*this_frame.width/640;
  Mat6x7* d_A; //Note, the 6x6 A and 6x1 b are combined into a single array so they can be reduced together with thrust later
  hipMalloc((void**)&d_A, (num_correspondences/load_size) * sizeof(Mat6x7));
  computeICPCostsUncorrespondedKernel << <(num_correspondences / 16 / load_size) + 1, 16 >> >(last_frame->normal, last_frame->vertex, this_frame.normal, this_frame.vertex, num_correspondences, load_size, d_A);
  hipDeviceSynchronize();

  //Sum terms (reduce) with thrust
  thrust::device_ptr<Mat6x7> thrust_A = thrust::device_pointer_cast<Mat6x7>(d_A);
  Mat6x7 matA = thrust::reduce(thrust_A, thrust_A + (num_correspondences/load_size));

  //Free up device memory
  hipFree(d_A);

  //Copy result to output
  memcpy(A, matA.values, 36 * sizeof(float));
  memcpy(b, matA.values + 36 , 6 * sizeof(float));
}

extern "C" void computeRGBDCost(const RGBDFrame* last_frame, const RGBDFrame& this_frame, float* A, float* b) {
  //TODO: Stuff here
  hipDeviceSynchronize();
}

} // namespace sensor

} // namespace octree_slam
