
// CUDA Dependencies
#include <hip/hip_runtime.h>

// Octree-SLAM Dependencies
#include <octree_slam/timing_utils.h>

//Global data
hipEvent_t beginEvent, endEvent;

void startTiming() {
	//Add timing options
	hipEventCreate( &beginEvent );
	hipEventCreate( &endEvent );

	//Execute the naive prefix sum and compute the time (in milliseconds)
	hipEventRecord(beginEvent, 0);
}

float stopTiming() {
	float time;

	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&time, beginEvent, endEvent);

	//Cleanup timers
	hipEventDestroy(beginEvent);
	hipEventDestroy(endEvent);

	return time;
}