
// CUDA Dependencies
#include <hip/hip_runtime.h>

// Octree-SLAM Dependencies
#include <octree_slam/common_types.h>

RawFrame::RawFrame(const int w, const int h) :
width(w), height(h) {
  hipMalloc((void**)&color, h*w*sizeof(Color256));
  hipMalloc((void**)&depth, h*w*sizeof(uint16_t));
}

RawFrame::~RawFrame() {
  hipFree(color);
  hipFree(depth);
}
