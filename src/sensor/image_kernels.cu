#include "hip/hip_runtime.h"

// Octree-SLAM Dependencies
#include <octree_slam/sensor/image_kernels.h>

// CUDA / OpenGL Dependencies
#include <cuda_gl_interop.h>

namespace octree_slam {

namespace sensor {

#define PI 3.14159

int GAUSS_RADIUS = 2;
float GAUSS_SIGMA = 100.0;
int BILATERAL_RADIUS = 2;
float BILATERAL_SIGMA = 100.0;
float3 INTENSITY_RATIO = { 0.299f, 0.587f, 0.114f }; //These are taken from Kintinuous

__global__ void generateVertexMapKernel(const uint16_t* depth_pixels, glm::vec3* vertex_map, const int width, const int height, const glm::vec2 focal_length) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  //Don't do anything if the index is out of bounds
  if (idx >= width*height) {
    return;
  }

  //Compute the x/y coords of this thread
  int x = idx % width;
  int y = idx / width;

  //Get the depth value for this pixel from global memory once
  int depth = depth_pixels[idx];
  //TODO: Handle no-measurements

  //Conversion from millimeters to meters
  const float milli = 0.001f;

  //Compute the point coordinates
  vertex_map[idx].x = (x - width/2) * (float) depth / focal_length.x * milli;
  vertex_map[idx].y = (height/2 - y) * (float) depth / focal_length.y * milli;
  vertex_map[idx].z = depth*milli;

}

extern "C" void generateVertexMap(const uint16_t* depth_pixels, glm::vec3* vertex_map, const int width, const int height, const glm::vec2 focal_length) {
  generateVertexMapKernel<<<width*height / 256 + 1, 256>>>(depth_pixels, vertex_map, width, height, focal_length);
  hipDeviceSynchronize();
}

__global__ void generateNormalMapKernel(const glm::vec3* vertex_map, glm::vec3* normal_map, const int width, const int height) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  //Don't do anything if the index is out of bounds
  if (idx >= width*height) {
    return;
  }

  //Compute the x/y coords of this thread
  int x = idx % width;
  int y = idx / width;

  //Get the center point from global memory once
  glm::vec3 center = vertex_map[idx];

  //Determine which direction to offset
  int offx = x > width/2 ? -1 : 1;
  int offy = y > height/2 ? -width : width;

  //Compute two vectors within the surface (locally)
  glm::vec3 v1 = vertex_map[idx + offx] - center;
  glm::vec3 v2 = vertex_map[idx + offy] - center;

  //Compute the normal
  glm::vec3 normal = glm::normalize( glm::cross(v1, v2) );

  //Store the result in global memory
  normal_map[idx] = normal;
}

extern "C" void generateNormalMap(const glm::vec3* vertex_map, glm::vec3* normal_map, const int width, const int height) {
  generateNormalMapKernel<<<width*height / 256 + 1, 256>>>(vertex_map, normal_map, width, height);
  hipDeviceSynchronize();
}

__host__ __device__ float gaussian1d(float x, float sigma) {
  float variance = pow(sigma, 2);
  float exponent = -pow(x, 2) / (2 * variance);
  return expf(exponent) / sqrt(2 * PI * variance);
}

__host__ __device__ uint2 idx_to_co(unsigned int idx, uint2 dims) {
  uint2 res;
  res.x = idx % dims.x;
  res.y = idx / dims.x;
  return res;
}

__host__ __device__ unsigned int co_to_idx(uint2 co, uint2 dims) {
  unsigned int res;
  res = co.y * dims.x + co.x;
  return res;
}

//This is borrowed from http://cs.au.dk/~staal/dpc/20072300_paper_final.pdf
__global__ void bilateralFilterGPU_v2(const uint16_t* input, uint16_t* output, uint2 dims, int radius, float* kernel, float sigma_range) {
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

  //Don't do anything if the index is out of bounds
  if (idx >= dims.x*dims.y) {
    return;
  }

  uint2 pos = idx_to_co(idx, dims);
  float currentColor = input[idx];
  float res = 0;
  float normalization = 0;

  for (int i = -radius; i <= radius; i++) {
    for (int j = -radius; j <= radius; j++) {
      int x_sample = pos.x + i;
      int y_sample = pos.y + j;

      //mirror edges
      if (x_sample < 0) x_sample = -x_sample;
      if (y_sample < 0) y_sample = -y_sample;
      if (x_sample > dims.x - 1) x_sample = dims.x - 1 - i;
      if (y_sample > dims.y - 1) y_sample = dims.y - 1 - j;
      float tmpColor =
        input[co_to_idx(make_uint2(x_sample, y_sample), dims)];

      //Don't continue if its a bad pixel
      if (tmpColor == 0 || tmpColor == 65535) {
        continue;
      }
      float gauss_spatial =
        kernel[co_to_idx(make_uint2(i + radius, j + radius), make_uint2(radius *
        2 + 1, radius * 2 + 1))];
      float gauss_range;
      gauss_range = gaussian1d(currentColor - tmpColor,
        sigma_range);
      float weight = gauss_spatial*gauss_range;
      normalization = normalization + weight;
      res = res + (tmpColor * weight);
    }
  }
  res /= normalization;

  output[idx] = res;
}

//This is borrowed and is used to compute a gaussian function on CPU or GPU
__host__ __device__ float gaussian2d(float x, float y, float sigma) {
  float variance = pow(sigma, 2);
  float exponent = -(pow(x, 2) + pow(y, 2)) / (2 * variance);
  return expf(exponent) / (2 * PI * variance);
}

//This is similarly borrowed to precompute the gaussian on the CPU
float* generateGaussianKernel(int radius, float sigma) {
  int area = (2 * radius + 1)*(2 * radius + 1);
  float* res = new float[area];
  for (int x = -radius; x <= radius; x++)
    for (int y = -radius; y <= radius; y++)
    {
    //Co_to_idx inspired
    int position = (x + radius)*(radius * 2 + 1) + y + radius;
    res[position] = gaussian2d(x, y, sigma);
    }
  return res;
}

/*
//TODO: Adapt to using texture memory so this more efficient implementation can be used
__global__ void bilateralFilterGPU_v5(float3* output, uint2 dims, int radius, float* kernel, float variance, float sqrt_pi_variance) {
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
  uint2 pos = idx_to_co(idx, dims);
  if (pos.x >= dims.x || pos.y >= dims.y) return;
  float3 currentColor = make_float3(tex1Dfetch(tex,
    3 * idx), tex1Dfetch(tex, 3 * idx + 1), tex1Dfetch(tex, 3 * idx + 2));
  float3 res = make_float3(0.0f, 0.0f, 0.0f);
  float3 normalization = make_float3(0.0f, 0.0f, 0.0f);
  float3 weight;
  for (int i = -radius; i <= radius; i++) {
    for (int j = -radius; j <= radius; j++) {
      int x_sample = pos.x + i;
      int y_sample = pos.y + j;
      //mirror edges
      if (x_sample < 0) x_sample = -x_sample;
      if (y_sample < 0) y_sample = -y_sample;
      if (x_sample > dims.x - 1) x_sample = dims.x - 1 - i;
      if (y_sample > dims.y - 1) y_sample = dims.y - 1 - j;
      int tempPos =
        co_to_idx(make_uint2(x_sample, y_sample), dims);
      float3 tmpColor = make_float3(tex1Dfetch(tex,
        3 * tempPos), tex1Dfetch(tex, 3 * tempPos + 1), tex1Dfetch(tex,
        3 * tempPos + 2));//input[tempPos];
      float gauss_spatial =
        kernel[co_to_idx(make_uint2(i + radius, j + radius), make_uint2(radius *
        2 + 1, radius * 2 + 1))];
      weight.x = gauss_spatial *
        gaussian1d_gpu_reg((currentColor.x -
        tmpColor.x), variance, sqrt_pi_variance);
      weight.y = gauss_spatial *
        gaussian1d_gpu_reg((currentColor.y -
        tmpColor.y), variance, sqrt_pi_variance);
      weight.z = gauss_spatial *
        gaussian1d_gpu_reg((currentColor.z -
        tmpColor.z), variance, sqrt_pi_variance);
      normalization = normalization + weight;
      res = res + (tmpColor * weight);
    }
  }
  res.x /= normalization.x;
  res.y /= normalization.y;
  res.z /= normalization.z;
  output[idx] = res;
}
*/

extern "C" void bilateralFilter(const uint16_t* depth_in, uint16_t* filtered_out, const int width, const int height) {
  //Create the gaussian kernel and transfer to GPU memory
  float* kernel = generateGaussianKernel(BILATERAL_RADIUS, 10.0);
  float* d_kernel;
  hipMalloc((void**)&d_kernel, (BILATERAL_RADIUS * 2 + 1)*(BILATERAL_RADIUS * 2 + 1)*sizeof(float));
  hipMemcpy(d_kernel, kernel, (BILATERAL_RADIUS * 2 + 1)*(BILATERAL_RADIUS * 2 + 1)*sizeof(float), hipMemcpyHostToDevice);
  delete kernel;

  //Use the bilateral filter kernel on the inputs
  uint2 dims = make_uint2(width, height);
  bilateralFilterGPU_v2<<<width*height/256 + 1, 256>>>(depth_in, filtered_out, dims, BILATERAL_RADIUS, d_kernel, BILATERAL_SIGMA);
  hipDeviceSynchronize();
  hipFree(d_kernel);
}

__global__ void colorToIntensityKernel(const Color256* color_in, float* intensity_out, const int size, const float3 intensity_ratio) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  //Don't do anything if the index is out of bounds
  if (idx >= size) {
    return;
  }

  intensity_out[idx] = color_in[idx].r/255.0f * intensity_ratio.x + color_in[idx].b/255.0f * intensity_ratio.y 
    + color_in[idx].b/255.0f * intensity_ratio.z;
}

extern "C" void colorToIntensity(const Color256* color_in, float* intensity_out, const int size) {
  colorToIntensityKernel<<<size/256 + 1, 256>>>(color_in, intensity_out, size, INTENSITY_RATIO);
  hipDeviceSynchronize();
}


__global__ void transformVertexMapKernel(glm::vec3* vertex, const glm::mat4 trans, const int size, const int load_size) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  //Don't do anything if the index is out of bounds
  if (idx*load_size >= size) {
    return;
  }

  //Determine whether the full load is in the bounds
  int bound = load_size;
  if ((idx+1)*load_size - size > 0 ) {
    bound -= (idx + 1)*load_size - size;
  }

  for (size_t i = 0; i < bound; i++) {
    vertex[load_size*idx + i] = glm::vec3(trans*glm::vec4(vertex[load_size*idx + i], 1.0f));
  }
}

extern "C" void transformVertexMap(glm::vec3* vertex_map, const glm::mat4 &trans, const int size) {
  int load_size = 16;
  transformVertexMapKernel<<<size / 256 / load_size + 1, 256>>>(vertex_map, trans, size, load_size);
}

__global__ void transformNormalMapKernel(glm::vec3* normal, const glm::mat4 trans, const int size, const int load_size) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  //Don't do anything if the index is out of bounds
  if (idx*load_size >= size) {
    return;
  }

  //Determine whether the full load is in the bounds
  int bound = load_size;
  if ((idx + 1)*load_size - size > 0) {
    bound -= (idx + 1)*load_size - size;
  }

  for (size_t i = 0; i < bound; i++) {
    normal[load_size*idx + i] = glm::vec3(trans*glm::vec4(normal[load_size*idx + i], 0.0f));
  }
}

extern "C" void transformNormalMap(glm::vec3* normal_map, const glm::mat4 &trans, const int size) {
  int load_size = 16;
  transformNormalMapKernel<<<size / 256 / load_size + 1, 256>>>(normal_map, trans, size, load_size);
}

template <class T>
__global__ void gaussianFilterKernel(const T* input, T* output, uint2 dims, int radius, float* kernel) {
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
  uint2 pos = idx_to_co(idx, dims);
  int img_x = pos.x;
  int img_y = pos.y;
  if (img_x >= dims.x || img_y >= dims.y) return;
  float res = 0;
  float normalization = 0;
  for (int i = -radius; i <= radius; i++) {
    for (int j = -radius; j <= radius; j++) {
      int x_sample = img_x + i;
      int y_sample = img_y + j;
      //mirror edges
      if (x_sample < 0) x_sample = -x_sample;
      if (y_sample < 0) y_sample = -y_sample;
      if (x_sample > dims.x - 1) x_sample = dims.x - 1 - i;
      if (y_sample > dims.y - 1) y_sample = dims.y - 1 - j;
      float tmpColor =
        input[co_to_idx(make_uint2(x_sample, y_sample), dims)];
      float gauss_spatial =
        kernel[co_to_idx(make_uint2(i + radius, j + radius), make_uint2(radius *
        2 + 1, radius * 2 + 1))];
      normalization = normalization + gauss_spatial;
      res = res + (tmpColor * gauss_spatial);
    }
  }
  res /= normalization;
  output[idx] = res;
}

template <class T>
void gaussianFilter(T* data, const int width, const int height) {
  //Create the gaussian kernel and transfer to GPU memory
  float* kernel = generateGaussianKernel(GAUSS_RADIUS, GAUSS_SIGMA);
  float* d_kernel;
  hipMalloc((void**)&d_kernel, (2*GAUSS_RADIUS+1)*(2*GAUSS_RADIUS+1)*sizeof(float));
  hipMemcpy(d_kernel, kernel, (2*GAUSS_RADIUS+1)*(2*GAUSS_RADIUS+1)*sizeof(float), hipMemcpyHostToDevice);
  delete kernel;

  //Create new memory space (this can't actually be done in place)
  T* data_new;
  hipMalloc((void**)&data_new, width*height*sizeof(T));

  //Use the bilateral filter kernel on the inputs
  uint2 dims = make_uint2(width, height);
  gaussianFilterKernel<<<width*height / 256 + 1, 256>>>(data, data_new, dims, GAUSS_RADIUS, d_kernel);
  hipDeviceSynchronize();

  //Copy into the input
  hipMemcpy(data, data_new, width*height*sizeof(T), hipMemcpyDeviceToDevice);

  //Free the temporary memory slot
  hipFree(data_new);
  hipFree(d_kernel);
}

//template void gaussianFilter<Color256>(Color256* data, const int width, const int height);
template void gaussianFilter<uint16_t>(uint16_t* data, const int width, const int height);
template void gaussianFilter<float>(float* data, const int width, const int height);

template <class T>
__global__ void subsampleKernel(const T* data_in, T* data_out, const int width, const int height) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  //Don't do anything if the index is out of bounds
  if (idx >= width*height) {
    return;
  }

  //Compute the x/y coords of this thread
  int x = idx % width;
  int y = idx / width;

  //Sample the value
  data_out[y*width + x] = data_in[4*y*width + 2*x];
}

template <class T>
void subsample(T* data, const int width, const int height) {
  //Create new memory space (this can't actually be done in place)
  T* data_new;
  hipMalloc((void**)&data_new, width*height*sizeof(T));

  subsampleKernel<<<width*height/1024 + 1, 256>>>(data, data_new, width/2, height/2);
  hipDeviceSynchronize();

  //Copy into the input
  hipMemcpy(data, data_new, width*height*sizeof(T), hipMemcpyDeviceToDevice);

  //Free the temporary memory slot
  hipFree(data_new);
}

//Declare types to generate symbols
template void subsample<Color256>(Color256* data, const int width, const int height);
template void subsample<uint16_t>(uint16_t* data, const int width, const int height);
template void subsample<float>(float* data, const int width, const int height);

} // namespace sensor

} // namespace octree_slam
